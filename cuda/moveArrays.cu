// moveArrays.cu
//      From Dr Dobbs tutorial: http://drdobbs.com/parallel/207200659?pgno=2

#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    float *a_h, *b_h;
    float *a_d, *b_d;
    int N = 14;
    int i;

    a_h = (float *)malloc(sizeof(float)*N);
    b_h = (float *)malloc(sizeof(float)*N);
    
    hipMalloc((void **) &a_d, sizeof(float)*N);
    hipMalloc((void **) &b_d, sizeof(float)*N);

    //initialize host data
    for (i=0; i<N; i++) {
        a_h[i] = 10.f+i;
        b_h[i] = 0.f;
    }

    // send data from host to device
    hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
    // copy data within device
    hipMemcpy(b_d, a_d, sizeof(float)*N, hipMemcpyDeviceToDevice);
    // retrieve data from device: b_d to b_h
    hipMemcpy(b_h, b_d, sizeof(float)*N, hipMemcpyDeviceToHost);

    // check result
    for (i=0; i<N; i++) 
        assert(a_h[i]==b_h[i]);

    free(a_h); free(b_h);
    hipFree(a_d); hipFree(b_d);
}
