#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// declaration of function to check for errors
void checkCudaError(const char* msg);

// kernel
__global__ void reverseArrayBlock(int *d_out, int *d_in) {
    int inOffset = blockDim.x * blockIdx.x;
    int outOffset = blockDim.x * (gridDim.x - 1 - blockIdx.x);
    int in = inOffset + threadIdx.x;
    int out = outOffset + (blockDim.x - 1 - threadIdx.x);
    d_out[out] = d_in[in];
}

// main thread
int main(int argc, char** argv) {
    int *h_a;
    int dimA = 256 * 1024;

    int *d_b, *d_a;

    int numThreadsPerBlock = 256;

    // compute number of blocks needed
    int numBlocks = dimA / numThreadsPerBlock;

    // allocate memory
    size_t memSize = numBlocks * numThreadsPerBlock*sizeof(int);
    h_a = (int *)malloc(memSize);
    hipMalloc((void **) &d_a, memSize);
    hipMalloc((void **) &d_b, memSize);

    // init array on host
    for (int i=0; i<dimA; i++) {
        h_a[i] = i;
    }

    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice);

    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArrayBlock <<< dimGrid, dimBlock >>> (d_b, d_a);
    
    // block until synced
    hipDeviceSynchronize();

    checkCudaError("kernel invocation");

    hipMemcpy(h_a, d_b, memSize, hipMemcpyDeviceToHost);

    checkCudaError("memcpy");

    for (int i=0; i<dimA; i++) {
        assert( h_a[i] == dimA - 1 - i);
    }

    hipFree(d_a); hipFree(d_b);
    free(h_a);
    return 0;
}

void checkCudaError(const char* msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s : %s .\n", msg, hipGetErrorString(err) );
        exit(EXIT_FAILURE);
    }
}

