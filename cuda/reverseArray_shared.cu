#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// declaration of function to check for errors
void checkCudaError(const char* msg);

// kernel
__global__ void reverseArrayBlock(int *d_out, int *d_in) {
    extern __shared__ int s_data[];

    int inOffset = blockDim.x * blockIdx.x;
    int in = inOffset + threadIdx.x;

    // load one element per thread from device memory and
    // store in reverse order into temp shared memory
    s_data[blockDim.x - 1 - threadIdx.x] = d_in[in];

    // block until all threads have written to shared memory
    __syncthreads();

    // write from shared memory in order, but to reversed
    // block offset as before
    int outOffset = blockDim.x * (gridDim.x - 1 - blockIdx.x);
    int out = outOffset + threadIdx.x;
    d_out[out] = s_data[threadIdx.x];
}

// main thread
int main(int argc, char** argv) {
    int *h_a;
    int dimA = 256 * 1024;

    int *d_b, *d_a;

    int numThreadsPerBlock = 256;

    // compute number of blocks needed
    int numBlocks = dimA / numThreadsPerBlock;

    int sharedMemSize = numThreadsPerBlock * sizeof(int);

    // allocate memory
    size_t memSize = numBlocks * numThreadsPerBlock*sizeof(int);
    h_a = (int *)malloc(memSize);
    hipMalloc((void **) &d_a, memSize);
    hipMalloc((void **) &d_b, memSize);

    // init array on host
    for (int i=0; i<dimA; i++) {
        h_a[i] = i;
    }

    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice);

    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArrayBlock <<< dimGrid, dimBlock, sharedMemSize >>> (d_b, d_a);
    
    // block until synced
    hipDeviceSynchronize();

    checkCudaError("kernel invocation");

    hipMemcpy(h_a, d_b, memSize, hipMemcpyDeviceToHost);

    checkCudaError("memcpy");

    for (int i=0; i<dimA; i++) {
        assert( h_a[i] == dimA - 1 - i);
    }

    hipFree(d_a); hipFree(d_b);
    free(h_a);
    return 0;
}

void checkCudaError(const char* msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s : %s .\n", msg, hipGetErrorString(err) );
        exit(EXIT_FAILURE);
    }
}

