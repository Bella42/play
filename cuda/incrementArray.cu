// incrementArray.cu
//      Second program from Dr Dobbs tutorial. http://drdobbs.com/parallel/207402986

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

// Host kernel = increment each element by 1
void incOnHost(float *a, int N) {
    int i;
    for (i=0; i<N; i++) 
        a[i] = a[i] + 1.f;
}

// Device Kernel = increment each element by 1
__global__ void incOnDevice(float *a, int N) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx<N)
        a[idx] = a[idx] + 1.f;
}

// Main thread
int main(void) {
    float *a_h, *b_h;
    float *a_d;
    int i, N=10;
    size_t size = N*sizeof(float);

    a_h = (float *)malloc(size);
    b_h = (float *)malloc(size);

    hipMalloc((void **) &a_d, size);

    for (i=0; i<N; i++) 
        a_h[i] = (float)i;

    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    
    // do calculation on host
    incOnHost(a_h, N);

    // do calculation on device
    int blockSize = 4;
    int nBlocks = N/blockSize + (N%blockSize == 0?0:1);
    incOnDevice <<< nBlocks, blockSize >>> (a_d, N);

    // Retrieve results and compare
    hipMemcpy(b_h,a_d,size,hipMemcpyDeviceToHost);
    for (i=0; i<N; i++) {
        printf("value from host %d = %1.1f\n",i,a_h[i]);
        printf("value from device %d = %1.1f\n",i,b_h[i]);
        assert(a_h[i] == b_h[i]);
    }

    free(a_h); free(b_h); hipFree(a_d);
}
